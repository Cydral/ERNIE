#include "hip/hip_runtime.h"
#include "cuda_dlib_ext.cuh"

namespace dlib
{
    namespace cuda
    {
   // ----------------------------------------------------------------------------------------

        __global__ void _cuda_rms_normalize(
            float* dest,
            float* scale,
            const float* src,
            const float* gamma,
            float eps,
            size_t ns,
            size_t ks,
            size_t num
        )
        {
            for (auto nk : grid_stride_range_y(0, ns * ks))
            {
                const auto n = nk / ks;
                const auto k = nk % ks;
                const auto p_src = src + (n * ks + k) * num;
                const auto p_dest = dest + (n * ks + k) * num;
                
                float sum_squares = 0.0f;
                for (auto i : grid_stride_range(0, num))
                {
                    sum_squares += p_src[i] * p_src[i];
                }
                warp_reduce_sum(scale[n * ks + k], 1.0f / sqrtf(sum_squares / num + eps));
                __syncthreads();

                const float s = scale[n * ks + k];
                const float g = gamma[k];
                for (auto i : grid_stride_range(0, num))
                {
                    p_dest[i] = p_src[i] * s * g;
                }
            }
        }

        void rms_normalize(
            const float eps,
            resizable_tensor& dest,
            resizable_tensor& scale,
            const tensor& src,
            const tensor& gamma
        )
        {
            const long num = src.nr() * src.nc();
            DLIB_CASSERT(
                src.k() == gamma.k() &&
                eps > 0,
                "\ngamma.k():  " << gamma.k() <<
                "\nsrc.k():    " << src.k() <<
                "\neps:  " << eps
            );

            dest.copy_size(src);
            scale.set_size(src.num_samples(), src.k());
            
            launch_kernel(_cuda_rms_normalize, max_jobs(src.num_samples() * src.k()), 
                        dest.device(), scale.device(), src.device(), gamma.device(), eps, 
                        src.num_samples(), src.k(), num);
        }

   // ----------------------------------------------------------------------------------------

        __global__ void _cuda_rms_normalize_gradient(float* out, float* gg, const float* s, const float* gi, const float* scale, const float* g, float* dscale, float eps, size_t ns, size_t num)
        {
            for (auto n : grid_stride_range_y(0, ns))
            {
                float temp_dscale = 0;
                for (auto i : grid_stride_range(0, num))
                {
                    auto idx = n * num + i;
                    const float x_hat = s[idx] * scale[n];
                    gg[i] += gi[idx] * x_hat;

                    const float dx = gi[idx] * g[i];
                    temp_dscale += dx * s[idx] * -0.5 * scale[n] * scale[n] * scale[n];
                }
                warp_reduce_atomic_add(dscale[n], temp_dscale);
            }
            __syncthreads();

            for (auto n : grid_stride_range_y(0, ns))
            {
                for (auto i : grid_stride_range(0, num))
                {
                    auto idx = n * num + i;
                    const float dx = gi[idx] * g[i];
                    out[idx] += dx * scale[n] + dscale[n] * 2 * s[idx] / num;
                }
            }
        }

        void rms_normalize_gradient(
            const double eps,
            const tensor& gradient_input,
            const tensor& scale,
            const tensor& src,
            const tensor& gamma,
            tensor& src_grad,
            tensor& gamma_grad,
            tensor& dscale
        )
        {
            const long num = src.k() * src.nr() * src.nc();
            DLIB_CASSERT(src.num_samples() == scale.size());
            DLIB_CASSERT(src.k() == gamma.k());
            DLIB_CASSERT(src.nr() == gamma.nr());
            DLIB_CASSERT(src.nc() == gamma.nc());
            DLIB_CASSERT(have_same_dimensions(gradient_input, src));
            DLIB_CASSERT(have_same_dimensions(gradient_input, src_grad));
            DLIB_CASSERT(have_same_dimensions(gamma_grad, gamma));
            DLIB_CASSERT(eps > 0);

            gamma_grad = 0;
            dscale = 0;
            launch_kernel(_cuda_rms_normalize_gradient, max_jobs(num, src.num_samples()),
                src_grad.device(), gamma_grad.device(), src.device(),
                gradient_input.device(), scale.device(), gamma.device(),
                dscale.device(), eps, src.num_samples(), num);
        }
    }
}